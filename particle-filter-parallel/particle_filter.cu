#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

//number of particles to run in particle filter
#define N_PARTICLES 1000
//number of iterations of the particle filter
#define N_ITERATIONS 10
//how many particles to assign to each thread block
#define DIVISIONS 256 

typedef unsigned long long timestamp_t;
static timestamp_t get_timestamp();

int main(){

	//initialize landmarks
	struct LandmarkData map_landmarks;
	map_landmarks.size = 4;
   	map_landmarks.landmarks = new struct Landmark[map_landmarks.size];

	map_landmarks.landmarks[0].x = 20.0;
	map_landmarks.landmarks[0].y = 20.0;

	map_landmarks.landmarks[1].x = 80.0;
	map_landmarks.landmarks[1].y = 80.0;

	map_landmarks.landmarks[2].x = 20.0;
	map_landmarks.landmarks[2].y = 80.0;

	map_landmarks.landmarks[3].x = 80.0;
	map_landmarks.landmarks[3].y = 20.0;


	//ground truth robot
	struct Robot my_robot;
	struct SensorData my_robot_sensor_data;
	my_robot_sensor_data.size = 4;
	my_robot_sensor_data.sensor_readings = new struct SensorRead[map_landmarks.size];
	init_robot(&myRobot);
	//move robot and reassign to my_robot
	move_and_get_particle(&my_robot, 0.1, 5.0, &my_robot);
	sense(&my_robot, &map_landmarks, &my_robot_sensor_data); 
	
	//setup and generate particles parallelized (prediction)
	int NUM_BLOCKS = N_PARTICLES / DIVISIONS;
	int NUM_THREADS = DIVISIONS; 
	//cudamalloc d_particle list
	generate_particles <<<NUM_BLOCKS, NUM_THREADS>>> (d_particle_list, NUM_THREADS, NUM_PARTICLES);
	
	for(int i = 0; i < N_ITERATIONS; i++){
	// for particle filter iterations
		//move ground truth robot
		move_and_get_particle(&my_robot, 0.1, 5.0, &my_robot);
		sense(&my_robot, &map_landmarks, &my_robot_sensor_data);
		
		// do motion update	
		//cuda malloc and copy my robot measurement dat to device
		motion_update_parallel <<<NUM_BLOCKS, NUM_THREADS>>>(d_particle_list); 

		//do motion update on particles (prediction)
		float* measurements = new float[N_PARTICLES];
		float* d_measurements;
		hipMalloc(d_measurements, N, sizeof(float));
		//
	}

	
		

/*	
	hipMalloc( &d_matrix, row * col * sizeof(float));
	hipMalloc( &d_result, row * col * sizeof(float));
	timestamp_t t0 = get_timestamp();

	hipMemcpy(d_matrix, matrix, col * row * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, col * row * sizeof(float), hipMemcpyHostToDevice);

	matTran <<<row, col>>> (row, col, d_result, row, col, d_matrix);
		
	hipMemcpy(result, d_result, col * row * sizeof(float), hipMemcpyDeviceToHost);

	timestamp_t t1 = get_timestamp();

	double diff = (double)t1 - (double)t0;
	printf("RUNNING TIME: %f microsecond\n", diff);	
*/
	delete map_landmarks->landmarks [];
	return 0;
}

static timestamp_t get_timestamp(){
	struct timeval now;
	gettimeofday(&now, NULL);
	return now.tv_usec + (timestamp_t)now.tv_sec * 1000000;
}
